#include "hip/hip_runtime.h"
//
// Created by astokely on 5/10/23.
//
#include <numeric>
#include <limits>
#include <iostream>
#include "hedetniemi.h"


__global__

void correlationToAdjacencyKernel(
        float *A,
        const float *C,
        int n
) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        if (i == j) {
            A[i * n + j] = 0.0;
        } else if (C[i * n + j] == 0.0) {
            A[i * n + j] = INFINITY;
        } else {
            A[i * n + j] = (float) 1.0 / C[i * n + j];
        }
    }
}

__global__ void hedetniemiAllShortestPathLengthsKernel(
        const float *A,
        float *H,
        int n
) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        float cij = INFINITY;
        for (int k = 0; k < n; k++) {
            auto AikHkjSum = A[j * n + k] + H[k * n + i];
            if (AikHkjSum < cij) {
                cij = AikHkjSum;
            }
        }
        H[i * n + j] = cij;
    }
}

__global__ void
hedetniemiAllShortestPathsPart1Kernel(
        const float *A,
        float *H,
        float *Hi,
        int n,
        int *p
) {
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        float cij = INFINITY;
        for (int k = 0; k < n; k++) {
            auto AikHkjSum = A[j * n + k] + H[k * n + i];
            if (AikHkjSum < cij) {
                cij = AikHkjSum;
            }
        }
        Hi[p[0] * (n * n) + i * n + j] = H[i * n + j];
        H[i * n + j] = cij;
    }
}

__global__ void hedetniemiAllShortestPathsKernelPart2(
        const float *H,
        const float *Hi,
        const float *A,
        int *paths,
        int n,
        int maxPathLength
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n) {
        if (i == j) {
            // The path from a vertex to itself is always zero
            paths[j * n + i] = i;
        } else {
            float pathLength = H[i * n + j];
            int k = j;
            for (int p = maxPathLength - 1; p >= 0; --p) {
                for (int q = 0; q < n; ++q) {
                    if (Hi[p * (n * n) + i * n + q] + A[q * n + k] ==
                        pathLength) {
                        pathLength -= A[q * n + k];
                        k = q;
                        break;
                    }
                }
                paths[i * n * maxPathLength + j * maxPathLength +
                      p] = k;
            }
        }
    }
}

void netcalc::hedetniemiAllShortestPathLengthsGpu(
        CuArray<float> *A,
        CuArray<float> *H,
        int maxPathLength
) {
    H->fromCuArrayDeepCopy(
            A,
            0,
            A->m() - 1,
            A->m(),
            A->n()
    );
    int numNodes = A->n();
    A->allocateDevice();
    A->toDevice();
    H->allocateDevice();
    H->toDevice();
    int threadsPerBlock = 16;
    int blocksPerGrid =
            (numNodes + threadsPerBlock - 1) / threadsPerBlock;
    dim3 block(threadsPerBlock,
               threadsPerBlock);
    dim3 grid(blocksPerGrid,
              blocksPerGrid);
    void *hedetniemiAllShortestPathLengthsKernelArgs[] = {
            &A->device(),
            &H->device(),
            &numNodes
    };
    for (int p = 0; p < maxPathLength; p++) {
        hipLaunchKernel((void *) hedetniemiAllShortestPathLengthsKernel,
                         grid,
                         block,
                         hedetniemiAllShortestPathLengthsKernelArgs
        );
    }
    H->toHost();
}

void netcalc::hedetniemiAllShortestPathsGpu(
        CuArray<float> *A,
        CuArray<float> *H,
        CuArray<int> *paths,
        int maxPathLength
) {
    H->fromCuArrayDeepCopy(
            A,
            0,
            A->m() - 1,
            A->m(),
            A->n()
    );
    auto Hi = new CuArray<float>;
    Hi->init(
            A->m() * maxPathLength,
            A->n()
    );
    Hi->allocateDevice();
    Hi->toDevice();
    int numNodes = A->n();
    A->allocateDevice();
    A->toDevice();
    H->allocateDevice();
    H->toDevice();
    int threadsPerBlock = 16;
    int blocksPerGrid =
            (numNodes + threadsPerBlock - 1) / threadsPerBlock;
    dim3 block(threadsPerBlock,
               threadsPerBlock);
    dim3 grid(blocksPerGrid,
              blocksPerGrid);
    paths->init(
            1,
            numNodes * maxPathLength * numNodes
    );
    for (int _ = 0; _ < paths->size(); _++) {
        paths->host()[_] = -1;
    }
    paths->allocateDevice();
    paths->toDevice();
    auto p = new CuArray<int>;
    p->init(1,
            1);
    p->allocateDevice();
    p->toDevice();
    void *hedetniemiAllShortestPathsPart1KernelArgs[] = {
            &A->device(),
            &H->device(),
            &Hi->device(),
            &numNodes,
            &p->device(),
    };
    for (int _ = 0; _ < maxPathLength; _++) {
        hipLaunchKernel((void *) hedetniemiAllShortestPathsPart1Kernel,
                         grid,
                         block,
                         hedetniemiAllShortestPathsPart1KernelArgs
        );
        p->toHost();
        p->host()[0] = _ + 1;
        p->toDevice();
    }
    void *hedetniemiAllShortestPathsPart2KernelArgs[] = {
            &H->device(),
            &Hi->device(),
            &A->device(),
            &paths->device(),
            &numNodes,
            &maxPathLength
    };
    hipLaunchKernel((void *) hedetniemiAllShortestPathsKernelPart2,
                     grid,
                     block,
                     hedetniemiAllShortestPathsPart2KernelArgs
    );
    paths->toHost();
    H->toHost();
    H->deallocateDevice();
    paths->deallocateDevice();
    A->deallocateDevice();
    paths->deallocateDevice();
    Hi->toHost();
    delete Hi;
    delete p;
}

void netcalc::correlationToAdjacencyGpu(
        CuArray<float> *A,
        CuArray<float> *C,
        int n
) {
    A->allocateDevice();
    if (!C->allocatedDevice()) {
        C->allocateDevice();
    }
    C->toDevice();
    A->toDevice();
    int threadsPerBlock = 16;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    dim3 block(threadsPerBlock,
               threadsPerBlock);
    dim3 grid(blocksPerGrid,
              blocksPerGrid);
    auto kernel = correlationToAdjacencyKernel;
    void *args[] = {&A->device(), &C->device(), &n};
    hipLaunchKernel((void *) kernel,
                     grid,
                     block,
                     args);
    A->toHost();
}